#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include <iostream>
#include <random>
#include <ctime>
#define N 10
using namespace std;



__global__ void myKernel(void) {

}

__global__ void add(int *a, int *b, int *c) {
    //threads can use shared memory within a block. using keyword __shared__
    __shared__ int temp;
    
    int x_idx = threadIdx.x +blockIdx.x*blockDim.x;
    int y_idx = threadIdx.y +blockIdx.y*blockDim.y;
    int z_idx = threadIdx.z +blockIdx.z*blockDim.z;
    *c = *a + *b;
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void cudaFunc() {
    srand(time(NULL));
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N*sizeof(int);

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = new int[N] {1};
    b = new int[N] {2};
    c = new int[N]{};

    // target, source, size, type
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);


    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    for (int i = 0; i<N; i++ ) {
        cout << a[i] << "+" << b[i] << "=" << c[i] << endl;
    }
    
}